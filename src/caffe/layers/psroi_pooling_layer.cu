#include "hip/hip_runtime.h"
// ------------------------------------------------------------------
// R-FCN
// Copyright (c) 2016 Microsoft
// Licensed under The MIT License [see r-fcn/LICENSE for details]
// Written by Yi Li
// ------------------------------------------------------------------

#include <cfloat>

#include "caffe/rfcn_layers.hpp"
#include "caffe/util/gpu_util.cuh"

using std::max;
using std::min;

namespace caffe {

  template <typename Dtype>
  __global__ void PSROIPoolingForward(
    const int nthreads,
    const Dtype* bottom_data,
    const Dtype spatial_scale,
    const int channels,
    const int height, const int width,
    const int pooled_height, const int pooled_width,
    const Dtype* bottom_rois,
    const int output_dim,
    const int group_size,
    Dtype* top_data,
    int* mapping_channel) {
    CUDA_KERNEL_LOOP(index, nthreads) {
      // The output is in order (n, ctop, ph, pw)
      int pw = index % pooled_width;
      int ph = (index / pooled_width) % pooled_height;
      int ctop = (index / pooled_width / pooled_height) % output_dim;
      int n = index / pooled_width / pooled_height / output_dim;

      // [start, end) interval for spatial sampling
      bottom_rois += n * 5;
      int roi_batch_ind = bottom_rois[0];
      Dtype roi_start_w = static_cast<Dtype>(round(bottom_rois[1])) * spatial_scale;
      Dtype roi_start_h = static_cast<Dtype>(round(bottom_rois[2])) * spatial_scale;
      Dtype roi_end_w = static_cast<Dtype>(round(bottom_rois[3]) + 1.) * spatial_scale;
      Dtype roi_end_h = static_cast<Dtype>(round(bottom_rois[4]) + 1.) * spatial_scale;

      // Force too small ROIs to be 1x1
      Dtype roi_width = max(roi_end_w - roi_start_w, 0.1); //avoid 0
      Dtype roi_height = max(roi_end_h - roi_start_h, 0.1);

      // Compute w and h at bottom 
      Dtype bin_size_h = roi_height / static_cast<Dtype>(pooled_height);
      Dtype bin_size_w = roi_width / static_cast<Dtype>(pooled_width);

      int hstart = floor(static_cast<Dtype>(ph) * bin_size_h
                          + roi_start_h);
      int wstart = floor(static_cast<Dtype>(pw)* bin_size_w
                          + roi_start_w);
      int hend = ceil(static_cast<Dtype>(ph + 1) * bin_size_h
                        + roi_start_h);
      int wend = ceil(static_cast<Dtype>(pw + 1) * bin_size_w
                        + roi_start_w);
      // Add roi offsets and clip to input boundaries
      hstart = min(max(hstart, 0), height);
      hend = min(max(hend, 0), height);
      wstart = min(max(wstart, 0),width);
      wend = min(max(wend, 0), width);
      bool is_empty = (hend <= hstart) || (wend <= wstart);

      int gw = pw;
      int gh = ph;
      int c = (ctop*group_size + gh)*group_size + gw;

      bottom_data += (roi_batch_ind * channels + c) * height * width;
      Dtype out_sum = 0;
      for (int h = hstart; h < hend; ++h){
        for (int w = wstart; w < wend; ++w){
          int bottom_index = h*width + w;
          out_sum += bottom_data[bottom_index];
        }
      }

      Dtype bin_area = (hend - hstart)*(wend - wstart);
      top_data[index] = is_empty? 0. : out_sum/bin_area;
      mapping_channel[index] = c;
    }
  }

  template <typename Dtype>
  __global__ void PSROIPoolingForward_RoiAlign(
      const int nthreads,
      const Dtype* bottom_data,
      const Dtype spatial_scale,
      const int channels,
      const int height, const int width,
      const int pooled_height, const int pooled_width,
      const Dtype* bottom_rois,
      const int output_dim,
      const int group_size,
      Dtype* top_data,
      int* mapping_channel) {
      CUDA_KERNEL_LOOP(index, nthreads) {
          // The output is in order (n, ctop, ph, pw)
          int pw = index % pooled_width;
          int ph = (index / pooled_width) % pooled_height;
          int ctop = (index / pooled_width / pooled_height) % output_dim;
          int n = index / pooled_width / pooled_height / output_dim;

          // [start, end) interval for spatial sampling
          bottom_rois += n * 5;
          int roi_batch_ind = bottom_rois[0];
          Dtype roi_start_w = bottom_rois[1] * spatial_scale;
          Dtype roi_start_h = bottom_rois[2] * spatial_scale;
          Dtype roi_end_w = bottom_rois[3] * spatial_scale;
          Dtype roi_end_h = bottom_rois[4] * spatial_scale;

          Dtype roi_width = roi_end_w - roi_start_w;
          Dtype roi_height = roi_end_h - roi_start_h;

          // Compute w and h at bottom
          Dtype bin_size_h = roi_height / static_cast<Dtype>(pooled_height);
          Dtype bin_size_w = roi_width / static_cast<Dtype>(pooled_width);

          // bin size is ratio of scaled roi height and 7,14, so it would be around 1,2,3

          //ph, pw is the position in the roi

          //therefore, we need use, bin size and ph to get the position of the 4 feature values

          int x1, x2, y1, y2;
          float px, py, pxmax, pymax, pxmin, pymin;
          pxmax = min(max(roi_start_w + static_cast<Dtype>(pw + 0.75) * bin_size_w, 0.001), width - 1.001);
          pymax = min(max(roi_start_h + static_cast<Dtype>(ph + 0.75) * bin_size_h, 0.001), height - 1.001);
          pxmin = min(max(roi_start_w + static_cast<Dtype>(pw + 0.25) * bin_size_w, 0.001), width - 1.001);
          pymin = min(max(roi_start_h + static_cast<Dtype>(ph + 0.25) * bin_size_h, 0.001), height - 1.001);

          Dtype out_sum = 0;
          int gw = pw;
          int gh = ph;
          int c = (ctop*group_size + gh)*group_size + gw;

          bottom_data += (roi_batch_ind * channels + c) * height * width;

          px = pxmin;
          py = pymin;

          x1 = floor(px);
          x2 = ceil(px);
          y1 = floor(py);
          y2 = ceil(py);

          out_sum += (px - x1)*(py - y1) * bottom_data[int(y2*width + x2)];
          out_sum += (px - x1)*(y2 - py) * bottom_data[int(y1*width + x2)];
          out_sum += (x2 - px)*(py - y1) * bottom_data[int(y2*width + x1)];
          out_sum += (x2 - px)*(y2 - py) * bottom_data[int(y1*width + x1)];

          px = pxmax;
          py = pymax;

          x1 = floor(px);
          x2 = ceil(px);
          y1 = floor(py);
          y2 = ceil(py);

          out_sum += (px - x1)*(py - y1) * bottom_data[int(y2*width + x2)];
          out_sum += (px - x1)*(y2 - py) * bottom_data[int(y1*width + x2)];
          out_sum += (x2 - px)*(py - y1) * bottom_data[int(y2*width + x1)];
          out_sum += (x2 - px)*(y2 - py) * bottom_data[int(y1*width + x1)];

          px = pxmin;
          py = pymax;

          x1 = floor(px);
          x2 = ceil(px);
          y1 = floor(py);
          y2 = ceil(py);

          out_sum += (px - x1)*(py - y1) * bottom_data[int(y2*width + x2)];
          out_sum += (px - x1)*(y2 - py) * bottom_data[int(y1*width + x2)];
          out_sum += (x2 - px)*(py - y1) * bottom_data[int(y2*width + x1)];
          out_sum += (x2 - px)*(y2 - py) * bottom_data[int(y1*width + x1)];

          px = pxmax;
          py = pymin;

          x1 = floor(px);
          x2 = ceil(px);
          y1 = floor(py);
          y2 = ceil(py);

          out_sum += (px - x1)*(py - y1) * bottom_data[int(y2*width + x2)];
          out_sum += (px - x1)*(y2 - py) * bottom_data[int(y1*width + x2)];
          out_sum += (x2 - px)*(py - y1) * bottom_data[int(y2*width + x1)];
          out_sum += (x2 - px)*(y2 - py) * bottom_data[int(y1*width + x1)];

          top_data[index] = out_sum / 4;
          mapping_channel[index] = c;
      }
  }

  template <typename Dtype>
  void PSROIPoolingLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
    const Dtype* bottom_data = bottom[0]->gpu_data();
    const Dtype* bottom_rois = bottom[1]->gpu_data();
    Dtype* top_data = top[0]->mutable_gpu_data();
    int* mapping_channel_ptr = mapping_channel_.mutable_gpu_data();
    int count = top[0]->count();
    caffe_gpu_set(count, Dtype(0), top_data);
    caffe_gpu_set(count, -1, mapping_channel_ptr);
    if (roi_align_ == false){
        // NOLINT_NEXT_LINE(whitespace/operators)
        PSROIPoolingForward<Dtype> << <CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS >> >(
            count, bottom_data, spatial_scale_, channels_, height_, width_, pooled_height_,
            pooled_width_, bottom_rois, output_dim_, group_size_, top_data, mapping_channel_ptr);
    }
    else {
        PSROIPoolingForward_RoiAlign<Dtype> << <CAFFE_GET_BLOCKS(count),
            CAFFE_CUDA_NUM_THREADS >> >(count, bottom_data, spatial_scale_,
            channels_, height_, width_, pooled_height_,
            pooled_width_, bottom_rois, output_dim_, group_size_,
            top_data, mapping_channel_ptr);
    }
    CUDA_POST_KERNEL_CHECK;

  }

  template <typename Dtype>
  __global__ void PSROIPoolingBackwardAtomic(
    const int nthreads,
    const Dtype* top_diff,
    const int* mapping_channel,
    const int num_rois,
    const Dtype spatial_scale,
    const int channels,
    const int height, const int width,
    const int pooled_height, const int pooled_width,
    const int output_dim, 
    Dtype* bottom_diff,
    const Dtype* bottom_rois) {
    CUDA_KERNEL_LOOP(index, nthreads) {
      // The output is in order (n, ctop, ph, pw)
      int pw = index % pooled_width;
      int ph = (index / pooled_width) % pooled_height;
      int n = index / pooled_width / pooled_height / output_dim;

      // [start, end) interval for spatial sampling
      bottom_rois += n * 5;
      int roi_batch_ind = bottom_rois[0];
      Dtype roi_start_w = static_cast<Dtype>(round(bottom_rois[1])) * spatial_scale;
      Dtype roi_start_h = static_cast<Dtype>(round(bottom_rois[2])) * spatial_scale;
      Dtype roi_end_w = static_cast<Dtype>(round(bottom_rois[3]) + 1.) * spatial_scale;
      Dtype roi_end_h = static_cast<Dtype>(round(bottom_rois[4]) + 1.) * spatial_scale;

      // Force too small ROIs to be 1x1
      Dtype roi_width = max(roi_end_w - roi_start_w, 0.1); //avoid 0
      Dtype roi_height = max(roi_end_h - roi_start_h, 0.1);

      // Compute w and h at bottom 
      Dtype bin_size_h = roi_height / static_cast<Dtype>(pooled_height);
      Dtype bin_size_w = roi_width / static_cast<Dtype>(pooled_width);

      int hstart = floor(static_cast<Dtype>(ph)* bin_size_h
        + roi_start_h);
      int wstart = floor(static_cast<Dtype>(pw)* bin_size_w
        + roi_start_w);
      int hend = ceil(static_cast<Dtype>(ph + 1) * bin_size_h
        + roi_start_h);
      int wend = ceil(static_cast<Dtype>(pw + 1) * bin_size_w
        + roi_start_w);
      // Add roi offsets and clip to input boundaries
      hstart = min(max(hstart, 0), height);
      hend = min(max(hend, 0), height);
      wstart = min(max(wstart, 0), width);
      wend = min(max(wend, 0), width);
      bool is_empty = (hend <= hstart) || (wend <= wstart);

      // Compute c at bottom
      int c = mapping_channel[index];
      Dtype* offset_bottom_diff = bottom_diff + (roi_batch_ind * channels + c) * height * width;
      Dtype bin_area = (hend - hstart)*(wend - wstart);
      Dtype diff_val = is_empty ? 0. : top_diff[index] / bin_area;
      for (int h = hstart; h < hend; ++h){
        for (int w = wstart; w < wend; ++w){
          int bottom_index = h*width + w;
          caffe_gpu_atomic_add(diff_val, offset_bottom_diff + bottom_index);
        }
      }
    }
  }

  template <typename Dtype>
  __global__ void PSROIPoolingBackwardAtomic_RoiAlign(
      const int nthreads,
      const Dtype* top_diff,
      const int* mapping_channel,
      const int num_rois,
      const Dtype spatial_scale,
      const int channels,
      const int height, const int width,
      const int pooled_height, const int pooled_width,
      const int output_dim,
      Dtype* bottom_diff,
      const Dtype* bottom_rois) {
      CUDA_KERNEL_LOOP(index, nthreads) {
          // The output is in order (n, ctop, ph, pw)
          int pw = index % pooled_width;
          int ph = (index / pooled_width) % pooled_height;
          int n = index / pooled_width / pooled_height / output_dim;

          // [start, end) interval for spatial sampling
          bottom_rois += n * 5;
          int roi_batch_ind = bottom_rois[0];
          Dtype roi_start_w = bottom_rois[1] * spatial_scale;
          Dtype roi_start_h = bottom_rois[2] * spatial_scale;
          Dtype roi_end_w = bottom_rois[3] * spatial_scale;
          Dtype roi_end_h = bottom_rois[4] * spatial_scale;

          Dtype roi_width = roi_end_w - roi_start_w;
          Dtype roi_height = roi_end_h - roi_start_h;

          // Compute w and h at bottom
          Dtype bin_size_h = roi_height / static_cast<Dtype>(pooled_height);
          Dtype bin_size_w = roi_width / static_cast<Dtype>(pooled_width);

          int x1, x2, y1, y2;
          float pxmin, pymin, pxmax, pymax, py, px;
          pxmax = min(max(roi_start_w + static_cast<Dtype>(pw + 0.75) * bin_size_w, 0.001), width - 1.001);
          pymax = min(max(roi_start_h + static_cast<Dtype>(ph + 0.75) * bin_size_h, 0.001), height - 1.001);
          pxmin = min(max(roi_start_w + static_cast<Dtype>(pw + 0.25) * bin_size_w, 0.001), width - 1.001);
          pymin = min(max(roi_start_h + static_cast<Dtype>(ph + 0.25) * bin_size_h, 0.001), height - 1.001);

          // Compute c at bottom
          int c = mapping_channel[index];
          Dtype* offset_bottom_diff = bottom_diff + (roi_batch_ind * channels + c) * height * width;
          Dtype diff_val = 0;
          diff_val = top_diff[index] / 4;

          px = pxmin;
          py = pymin;
          x1 = floor(px);
          x2 = ceil(px);
          y1 = floor(py);
          y2 = ceil(py);
          caffe_gpu_atomic_add(diff_val * (px - x1)*(py - y1), offset_bottom_diff + int(y2*width + x2));
          caffe_gpu_atomic_add(diff_val * (px - x1)*(y2 - py), offset_bottom_diff + int(y1*width + x2));
          caffe_gpu_atomic_add(diff_val * (x2 - px)*(py - y1), offset_bottom_diff + int(y2*width + x1));
          caffe_gpu_atomic_add(diff_val * (x2 - px)*(y2 - py), offset_bottom_diff + int(y1*width + x1));

          px = pxmax;
          py = pymax;
          x1 = floor(px);
          x2 = ceil(px);
          y1 = floor(py);
          y2 = ceil(py);
          caffe_gpu_atomic_add(diff_val * (px - x1)*(py - y1), offset_bottom_diff + int(y2*width + x2));
          caffe_gpu_atomic_add(diff_val * (px - x1)*(y2 - py), offset_bottom_diff + int(y1*width + x2));
          caffe_gpu_atomic_add(diff_val * (x2 - px)*(py - y1), offset_bottom_diff + int(y2*width + x1));
          caffe_gpu_atomic_add(diff_val * (x2 - px)*(y2 - py), offset_bottom_diff + int(y1*width + x1));

          px = pxmin;
          py = pymax;
          x1 = floor(px);
          x2 = ceil(px);
          y1 = floor(py);
          y2 = ceil(py);
          caffe_gpu_atomic_add(diff_val * (px - x1)*(py - y1), offset_bottom_diff + int(y2*width + x2));
          caffe_gpu_atomic_add(diff_val * (px - x1)*(y2 - py), offset_bottom_diff + int(y1*width + x2));
          caffe_gpu_atomic_add(diff_val * (x2 - px)*(py - y1), offset_bottom_diff + int(y2*width + x1));
          caffe_gpu_atomic_add(diff_val * (x2 - px)*(y2 - py), offset_bottom_diff + int(y1*width + x1));

          px = pxmax;
          py = pymin;
          x1 = floor(px);
          x2 = ceil(px);
          y1 = floor(py);
          y2 = ceil(py);
          caffe_gpu_atomic_add(diff_val * (px - x1)*(py - y1), offset_bottom_diff + int(y2*width + x2));
          caffe_gpu_atomic_add(diff_val * (px - x1)*(y2 - py), offset_bottom_diff + int(y1*width + x2));
          caffe_gpu_atomic_add(diff_val * (x2 - px)*(py - y1), offset_bottom_diff + int(y2*width + x1));
          caffe_gpu_atomic_add(diff_val * (x2 - px)*(y2 - py), offset_bottom_diff + int(y1*width + x1));
      }
  }

  template <typename Dtype>
  void PSROIPoolingLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
    if (!propagate_down[0]) {
      return;
    }

    const Dtype* bottom_rois = bottom[1]->gpu_data();
    const Dtype* top_diff = top[0]->gpu_diff();
    Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
    const int bottom_count = bottom[0]->count();
    const int* mapping_channel_ptr = mapping_channel_.gpu_data();
    caffe_gpu_set(bottom[1]->count(), Dtype(0), bottom[1]->mutable_gpu_diff());
    caffe_gpu_set(bottom_count, Dtype(0), bottom_diff);
    const int count = top[0]->count();
    if (roi_align_ == false){
        // NOLINT_NEXT_LINE(whitespace/operators)
        PSROIPoolingBackwardAtomic<Dtype> << <CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS >> >(
            count, top_diff, mapping_channel_ptr, top[0]->num(), spatial_scale_,
            channels_, height_, width_, pooled_height_, pooled_width_, output_dim_,
            bottom_diff, bottom_rois);
    }
    else {
        PSROIPoolingBackwardAtomic_RoiAlign<Dtype> << <CAFFE_GET_BLOCKS(count),
            CAFFE_CUDA_NUM_THREADS >> >(count, top_diff, mapping_channel_ptr,
            top[0]->num(), spatial_scale_, channels_, height_, width_,
            pooled_height_, pooled_width_, output_dim_, bottom_diff,
            bottom_rois);
    }

    CUDA_POST_KERNEL_CHECK;
  }

  INSTANTIATE_LAYER_GPU_FUNCS(PSROIPoolingLayer);

}  // namespace caffe
